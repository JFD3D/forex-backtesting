#include "hip/hip_runtime.h"
#include "studies/polynomialRegressionChannelStudy.cuh"

// Sources: https://rosettacode.org/wiki/Polynomial_regression#C, http://stackoverflow.com/a/36524956/83897
Real PolynomialRegressionChannelStudy::calculateRegression(std::vector<Real> &values, int degree) {
    gsl_multifit_linear_workspace *ws;
    gsl_matrix *cov, *X;
    gsl_vector *y, *c;
    double chisq;
    int obs;
    std::vector<double> coefficients;
    Real point;

    int i = 0;
    int j = 0;

    obs = values.size();

    X = gsl_matrix_alloc(obs, degree);
    y = gsl_vector_alloc(obs);
    c = gsl_vector_alloc(degree);
    cov = gsl_matrix_alloc(degree, degree);

    for (i=0; i<obs; i++) {
        for (j=0; j<degree; j++) {
            gsl_matrix_set(X, i, j, pow(i, j));
        }
        gsl_vector_set(y, i, values[i]);
    }

    ws = gsl_multifit_linear_alloc(obs, degree);
    gsl_multifit_linear(X, y, c, cov, &chisq, ws);

    // Get coefficients.
    for (i=0; i<degree; i++) {
        coefficients.push_back(gsl_vector_get(c, i));
    }

    gsl_multifit_linear_free(ws);
    gsl_matrix_free(X);
    gsl_matrix_free(cov);
    gsl_vector_free(y);
    gsl_vector_free(c);

    // Calculate the last data point in the series (that's all that is needed for
    // the purposes of this study).
    point = coefficients[0];
    for (i=1; i<degree; i++) {
        point += pow(obs - 1, i) * coefficients[i];
    }

    return point;
}

Real PolynomialRegressionChannelStudy::calculateStandardDeviation(std::vector<Real> &values) {
    Real sum = 0;
    Real squaredSum = 0;
    Real mean;
    Real variance;
    int valueCount = values.size();
    int i = 0;

    if (valueCount == 0) {
        return 0.0;
    }

    for (i=0; i<valueCount; ++i) {
        sum += values[i];
        squaredSum += values[i] * values[i];
    }

    mean = sum / valueCount;
    variance = (squaredSum / valueCount) - (mean * mean);

    return sqrt(variance);
}

void PolynomialRegressionChannelStudy::tick() {
    Tick *lastTick = getLastTick();
    std::vector<Tick*> *dataSegment = nullptr;
    int dataSegmentLength = 0;
    std::string regressionOutputName = getOutputMapping("regression");
    Real regression;
    Real regressionStandardDeviation;

    resetTickOutputs();

    dataSegment = getDataSegment(getInput("length"));
    dataSegmentLength = dataSegment->size();

    if (dataSegmentLength <= 1) {
        // Reset.
        std::vector<Real>().swap(this->pastPrices);
        std::vector<Real>().swap(this->pastRegressions);
    }

    // Record another past price.
    this->pastPrices.push_back(lastTick->at("close"));

    if (dataSegmentLength < getInput("length")) {
        setTickOutput(regressionOutputName, 0.0);
        setTickOutput(getOutputMapping("upper"), 0.0);
        setTickOutput(getOutputMapping("lower"), 0.0);

        delete dataSegment;
        return;
    }

    // Keep the correct number of past prices.
    while (this->pastPrices.size() > getInput("length")) {
        this->pastPrices.erase(this->pastPrices.begin());
    }

    // Calculate the regression.
    regression = calculateRegression(this->pastPrices, getInput("degree"));

    // Record another past regression.
    this->pastRegressions.push_back(regression);

    // Keep the correct number of past regressions.
    while (this->pastRegressions.size() > getInput("length")) {
        this->pastRegressions.erase(this->pastRegressions.begin());
    }

    // Calculate the standard deviation from the regressions.
    regressionStandardDeviation = calculateStandardDeviation(this->pastRegressions);

    setTickOutput(regressionOutputName, regression);

    // Calculate the upper and lower values.
    setTickOutput(getOutputMapping("upper"), regression + (regressionStandardDeviation * getInput("deviations")));
    setTickOutput(getOutputMapping("lower"), regression - (regressionStandardDeviation * getInput("deviations")));

    // Free memory.
    delete dataSegment;
}
