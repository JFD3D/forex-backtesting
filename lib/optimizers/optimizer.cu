#include "hip/hip_runtime.h"
#include "optimizers/optimizer.cuh"

__global__ void optimizer_initialize(thrust::device_vector<Strategy*> strategies, thrust::device_vector<Configuration*> configurations, int configurationCount) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < configurationCount) {
        // Set up one strategy instance per configuration.
        // strategies[i] = OptimizationStrategyFactory::create(strategyName, symbol, dataIndex, group, configurations[i]);
    }
}

__global__ void optimizer_backtest(
    thrust::device_vector<double*> data,
    thrust::device_vector<Strategy*> strategies,
    int dataPointIndex,
    int configurationCount,
    double investment,
    double profitability
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < configurationCount) {
        // strategies[i]->backtest(data[dataPointIndex], investment, profitability);
    }
}

Optimizer::Optimizer(mongoc_client_t *dbClient, const char *strategyName, const char *symbol, int group) {
    this->dbClient = dbClient;
    this->strategyName = strategyName;
    this->symbol = symbol;
    this->group = group;
    this->dataCount = 0;
    this->dataIndex = new std::map<std::string, int>();
}

bson_t *Optimizer::convertTickToBson(Tick *tick) {
    bson_t *document;
    bson_t dataDocument;

    document = bson_new();
    BSON_APPEND_UTF8(document, "symbol", this->symbol);
    BSON_APPEND_INT32(document, "testingGroups", tick->at("testingGroups"));
    BSON_APPEND_INT32(document, "validationGroups", tick->at("validationGroups"));
    BSON_APPEND_DOCUMENT_BEGIN(document, "data", &dataDocument);

    // Remove group keys as they are no longer needed.
    tick->erase("testingGroups");
    tick->erase("validationGroups");

    // Add tick properties to document.
    for (Tick::iterator propertyIterator = tick->begin(); propertyIterator != tick->end(); ++propertyIterator) {
        bson_append_double(&dataDocument, propertyIterator->first.c_str(), propertyIterator->first.length(), propertyIterator->second);
    }

    bson_append_document_end(document, &dataDocument);

    return document;
}

void Optimizer::saveTicks(std::vector<Tick*> ticks) {
    if (ticks.size() == 0) {
        return;
    }

    mongoc_collection_t *collection;
    mongoc_bulk_operation_t *bulkOperation;
    bson_t bulkOperationReply;
    bson_error_t bulkOperationError;

    // Get a reference to the database collection.
    collection = mongoc_client_get_collection(this->dbClient, "forex-backtesting-test", "datapoints");

    // Begin a bulk operation.
    bulkOperation = mongoc_collection_create_bulk_operation(collection, true, NULL);

    // Reference: http://api.mongodb.org/c/current/bulk.html
    for (std::vector<Tick*>::iterator insertionIterator = ticks.begin(); insertionIterator != ticks.end(); ++insertionIterator) {
        bson_t *document = convertTickToBson(*insertionIterator);
        mongoc_bulk_operation_insert(bulkOperation, document);
        bson_destroy(document);
    }

    // Execute the bulk operation.
    mongoc_bulk_operation_execute(bulkOperation, &bulkOperationReply, &bulkOperationError);

    // Cleanup.
    mongoc_collection_destroy(collection);
    mongoc_bulk_operation_destroy(bulkOperation);
    bson_destroy(&bulkOperationReply);
}

void Optimizer::prepareData(std::vector<Tick*> ticks) {
    double percentage;
    int tickCount = ticks.size();
    std::vector<Tick*> cumulativeTicks;
    int cumulativeTickCount;
    int threadCount = std::thread::hardware_concurrency();
    maginatics::ThreadPool pool(1, threadCount, 5000);
    std::vector<Study*> studies = this->getStudies();
    int i = 0;
    int j = 0;

    // Reserve space in advance for better performance.
    cumulativeTicks.reserve(tickCount);

    printf("Preparing data...");

    // Go through the data and run studies for each data item.
    for (std::vector<Tick*>::iterator tickIterator = ticks.begin(); tickIterator != ticks.end(); ++tickIterator) {
        // Show progress.
        percentage = (++i / (double)tickCount) * 100.0;
        printf("\rPreparing data...%0.4f%%", percentage);

        Tick *tick = *tickIterator;
        Tick *previousTick = nullptr;

        if (cumulativeTicks.size() > 0) {
            previousTick = cumulativeTicks.back();
        }

        // If the previous tick's minute was not the previous minute, then save the current
        // ticks, and start over with recording.
        if (previousTick && ((*tick)["timestamp"] - (*previousTick)["timestamp"]) > 60) {
            previousTick = nullptr;

            // Save and then remove the current cumulative ticks.
            saveTicks(cumulativeTicks);

            // Release memory.
            cumulativeTickCount = cumulativeTicks.size();
            for (j=0; j<cumulativeTickCount; j++) {
                delete cumulativeTicks[j];
                cumulativeTicks[j] = nullptr;
            }
            std::vector<Tick*>().swap(cumulativeTicks);
        }

        previousTick = tick;

        // Append to the cumulative data.
        cumulativeTicks.push_back(tick);

        for (std::vector<Study*>::iterator studyIterator = studies.begin(); studyIterator != studies.end(); ++studyIterator) {
            // Update the data for the study.
            (*studyIterator)->setData(&cumulativeTicks);

            // Use a thread pool so that all CPU cores can be used.
            pool.execute([studyIterator]() {
                // Process the latest data for the study.
                (*studyIterator)->tick();
            });
        }

        // Block until all tasks for the current data point complete.
        pool.drain();

        // Merge tick output values from the studies into the current tick.
        for (std::vector<Study*>::iterator studyIterator = studies.begin(); studyIterator != studies.end(); ++studyIterator) {
            std::map<std::string, double> studyOutputs = (*studyIterator)->getTickOutputs();

            for (std::map<std::string, double>::iterator outputIterator = studyOutputs.begin(); outputIterator != studyOutputs.end(); ++outputIterator) {
                (*tick)[outputIterator->first] = outputIterator->second;
            }
        }

        // Periodically save tick data to the database and free up memory.
        if (cumulativeTicks.size() >= 2000) {
            // Extract the first ~1000 ticks to be inserted.
            std::vector<Tick*> firstCumulativeTicks(cumulativeTicks.begin(), cumulativeTicks.begin() + (cumulativeTicks.size() - 1000));

            // Write ticks to database.
            saveTicks(firstCumulativeTicks);

            // Release memory.
            for (j=0; j<1000; j++) {
                delete cumulativeTicks[j];
                cumulativeTicks[j] = nullptr;
            }
            std::vector<Tick*>().swap(firstCumulativeTicks);

            // Keep only the last 1000 elements.
            std::vector<Tick*>(cumulativeTicks.begin() + (cumulativeTicks.size() - 1000), cumulativeTicks.end()).swap(cumulativeTicks);
        }

        tick = nullptr;
        previousTick = nullptr;
    }

    printf("\n");
}

int Optimizer::getDataPropertyCount() {
    std::vector<Study*> studies = this->getStudies();
    int basePropertyCount = 5;
    int propertyCount = basePropertyCount;

    for (std::vector<Study*>::iterator iterator = studies.begin(); iterator != studies.end(); ++iterator) {
        propertyCount += (*iterator)->getOutputMap().size();
    }

    return propertyCount;
}

void Optimizer::loadData() {
    printf("Loading data...");

    double percentage;
    int propertyIndex = 0;
    mongoc_collection_t *collection;
    mongoc_cursor_t *cursor;
    bson_t *countQuery;
    bson_t *query;
    const bson_t *document;
    bson_iter_t documentIterator;
    bson_iter_t dataIterator;
    bson_error_t error;
    const char *propertyName;
    const bson_value_t *propertyValue;
    int dataPropertyCount = this->getDataPropertyCount();
    int i = 0;

    // Get a reference to the database collection.
    collection = mongoc_client_get_collection(this->dbClient, "forex-backtesting-test", "datapoints");

    // Query for the number of data points.
    countQuery = BCON_NEW("symbol", BCON_UTF8(this->symbol));
    this->dataCount = mongoc_collection_count(collection, MONGOC_QUERY_NONE, countQuery, 0, 0, NULL, &error);

    if (this->dataCount < 0) {
        // No data points found.
        throw std::runtime_error(error.message);
    }

    // Query the database.
    query = BCON_NEW(
        "$query", "{", "symbol", BCON_UTF8(this->symbol), "}",
        "$orderby", "{", "data.timestamp", BCON_INT32(1), "}",
        "$hint", "{", "data.timestamp", BCON_INT32(1), "}"
    );
    cursor = mongoc_collection_find(collection, MONGOC_QUERY_NONE, 0, 0, 1000, query, NULL, NULL);

    // Go through query results, and convert each document into an array.
    while (mongoc_cursor_next(cursor, &document)) {
        double *dataPoint;
        propertyIndex = 0;

        // Allocate memory for the data point.
        dataPoint = (double*)malloc(dataPropertyCount * sizeof(double));

        if (bson_iter_init(&documentIterator, document)) {
            // Find the "data" subdocument.
            if (bson_iter_init_find(&documentIterator, document, "data") &&
                BSON_ITER_HOLDS_DOCUMENT(&documentIterator) &&
                bson_iter_recurse(&documentIterator, &dataIterator)) {

                // Iterate through the data properties.
                while (bson_iter_next(&dataIterator)) {
                    propertyValue = bson_iter_value(&dataIterator);

                    // Add the data property value to the data store.
                    dataPoint[propertyIndex] = propertyValue->value.v_double;

                    // For the first data point only (only need to do this once), build an
                    // index of data item positions.
                    if (this->dataCount == 0) {
                        // Get the property name.
                        propertyName = bson_iter_key(&dataIterator);

                        // Add to the data index map.
                        (*this->dataIndex)[propertyName] = propertyIndex;
                    }

                    propertyIndex++;
                }
            }
        }

        // Show progress.
        percentage = (++i / (double)this->dataCount) * 100.0;
        printf("\rLoading data...%0.4f%%", percentage);
    }

    printf("\n");

    // Cleanup.
    bson_destroy(countQuery);
    bson_destroy(query);
    mongoc_cursor_destroy(cursor);
    mongoc_collection_destroy(collection);
}

std::vector<MapConfiguration*> *Optimizer::buildMapConfigurations(
    std::map<std::string, ConfigurationOption> options,
    int optionIndex,
    std::vector<MapConfiguration*> *results,
    MapConfiguration *current
) {
    std::vector<std::string> allKeys;
    std::string optionKey;
    ConfigurationOption configurationOptions;

    // Get all options keys.
    for (std::map<std::string, ConfigurationOption>::iterator optionsIterator = options.begin(); optionsIterator != options.end(); ++optionsIterator) {
        allKeys.push_back(optionsIterator->first);
    }

    optionKey = allKeys[optionIndex];
    configurationOptions = options[optionKey];

    for (ConfigurationOption::iterator configurationOptionsIterator = configurationOptions.begin(); configurationOptionsIterator != configurationOptions.end(); ++configurationOptionsIterator) {
        // Iterate through configuration option values.
        for (std::map<std::string, boost::variant<std::string, double>>::iterator valuesIterator = configurationOptionsIterator->begin(); valuesIterator != configurationOptionsIterator->end(); ++valuesIterator) {
            if (valuesIterator->second.type() == typeid(std::string)) {
                // Value points to a key.
                (*current)[valuesIterator->first] = (*this->dataIndex)[boost::get<std::string>(valuesIterator->second)];
            }
            else {
                // Value is an actual value.
                (*current)[valuesIterator->first] = boost::get<double>(valuesIterator->second);
            }
        }

        if (optionIndex + 1 < allKeys.size()) {
            buildMapConfigurations(options, optionIndex + 1, results, current);
        }
        else {
            results->push_back(current);
        }
    }

    return results;
}

thrust::host_vector<Configuration*> Optimizer::buildConfigurations(std::map<std::string, ConfigurationOption> options) {
    printf("Building configurations...");

    std::vector<MapConfiguration*> *mapConfigurations = buildMapConfigurations(options);
    thrust::host_vector<Configuration*> configurations;
    Configuration *configuration = new Configuration();

    // Reserve space in advance for better performance.
    configurations.reserve(mapConfigurations->size());

    // Convert map representations of maps into structs of type Configuration.
    for (std::vector<MapConfiguration*>::iterator mapConfigurationIterator = mapConfigurations->begin(); mapConfigurationIterator != mapConfigurations->end(); ++mapConfigurationIterator) {
        // Set up a new, empty configuration.
        configuration = new Configuration();

        // Set basic properties.
        configuration->timestamp = (*this->dataIndex)["timestamp"];
        configuration->open = (*this->dataIndex)["open"];
        configuration->high = (*this->dataIndex)["high"];
        configuration->low = (*this->dataIndex)["low"];
        configuration->close = (*this->dataIndex)["close"];

        // Set index mappings.
        if ((*mapConfigurationIterator)->find("sma13") != (*mapConfigurationIterator)->end()) {
            configuration->sma13 = boost::get<int>((**mapConfigurationIterator)["sma13"]);
        }
        if ((*mapConfigurationIterator)->find("ema50") != (*mapConfigurationIterator)->end()) {
            configuration->ema50 = boost::get<int>((**mapConfigurationIterator)["ema50"]);
        }
        if ((*mapConfigurationIterator)->find("ema100") != (*mapConfigurationIterator)->end()) {
            configuration->ema100 = boost::get<int>((**mapConfigurationIterator)["ema100"]);
        }
        if ((*mapConfigurationIterator)->find("ema200") != (*mapConfigurationIterator)->end()) {
            configuration->ema200 = boost::get<int>((**mapConfigurationIterator)["ema200"]);
        }
        if ((*mapConfigurationIterator)->find("ema250") != (*mapConfigurationIterator)->end()) {
            configuration->ema250 = boost::get<int>((**mapConfigurationIterator)["ema250"]);
        }
        if ((*mapConfigurationIterator)->find("ema300") != (*mapConfigurationIterator)->end()) {
            configuration->ema300 = boost::get<int>((**mapConfigurationIterator)["ema300"]);
        }
        if ((*mapConfigurationIterator)->find("ema350") != (*mapConfigurationIterator)->end()) {
            configuration->ema350 = boost::get<int>((**mapConfigurationIterator)["ema350"]);
        }
        if ((*mapConfigurationIterator)->find("ema400") != (*mapConfigurationIterator)->end()) {
            configuration->ema400 = boost::get<int>((**mapConfigurationIterator)["ema400"]);
        }
        if ((*mapConfigurationIterator)->find("ema450") != (*mapConfigurationIterator)->end()) {
            configuration->ema450 = boost::get<int>((**mapConfigurationIterator)["ema450"]);
        }
        if ((*mapConfigurationIterator)->find("ema500") != (*mapConfigurationIterator)->end()) {
            configuration->ema500 = boost::get<int>((**mapConfigurationIterator)["ema500"]);
        }
        if ((*mapConfigurationIterator)->find("rsi") != (*mapConfigurationIterator)->end()) {
            configuration->rsi = boost::get<int>((**mapConfigurationIterator)["rsi"]);
        }
        if ((*mapConfigurationIterator)->find("stochasticD") != (*mapConfigurationIterator)->end()) {
            configuration->stochasticD = boost::get<int>((**mapConfigurationIterator)["stochasticD"]);
        }
        if ((*mapConfigurationIterator)->find("stochasticK") != (*mapConfigurationIterator)->end()) {
            configuration->stochasticK = boost::get<int>((**mapConfigurationIterator)["stochasticK"]);
        }
        if ((*mapConfigurationIterator)->find("prChannelUpper") != (*mapConfigurationIterator)->end()) {
            configuration->prChannelUpper = boost::get<int>((**mapConfigurationIterator)["prChannelUpper"]);
        }
        if ((*mapConfigurationIterator)->find("prChannelLower") != (*mapConfigurationIterator)->end()) {
            configuration->prChannelLower = boost::get<int>((**mapConfigurationIterator)["prChannelLower"]);
        }

        // Set values.
        if ((*mapConfigurationIterator)->find("rsiOverbought") != (*mapConfigurationIterator)->end()) {
            configuration->rsiOverbought = boost::get<double>((**mapConfigurationIterator)["rsiOverbought"]);
        }
        if ((*mapConfigurationIterator)->find("rsiOversold") != (*mapConfigurationIterator)->end()) {
            configuration->rsiOversold = boost::get<double>((**mapConfigurationIterator)["rsiOversold"]);
        }
        if ((*mapConfigurationIterator)->find("stochasticOverbought") != (*mapConfigurationIterator)->end()) {
            configuration->stochasticOverbought = boost::get<double>((**mapConfigurationIterator)["stochasticOverbought"]);
        }
        if ((*mapConfigurationIterator)->find("stochasticOversold") != (*mapConfigurationIterator)->end()) {
            configuration->stochasticOversold = boost::get<double>((**mapConfigurationIterator)["stochasticOversold"]);
        }

        configurations.push_back(configuration);
    }

    printf("%i configurations built\n", (int)configurations.size());

    return configurations;
}

void Optimizer::optimize(thrust::host_vector<Configuration*> &configurations, double investment, double profitability) {
    printf("Optimizing...");

    double percentage;
    int configurationCount = configurations.size();
    int dataChunkSize = 500000;
    int dataPointCount = this->data.size();
    int i = 0;

    // Host data.
    thrust::host_vector<Strategy*> strategies(configurationCount);

    // GPU settings.
    int blockCount = 32;
    int threadsPerBlock = 1024;

    // Copy data to the GPU.
    thrust::host_vector<double*> dataSegment;
    thrust::device_vector<double*> devDataSegment;
    thrust::device_vector<Strategy*> devStrategies = strategies;
    thrust::device_vector<Configuration*> devConfigurations = configurations;

    // Initialize strategies on the GPU.
    optimizer_initialize<<<blockCount, threadsPerBlock>>>(devStrategies, configurations, configurationCount);

    // Iterate over data ticks.
    for (i=0; i<this->dataCount; i++) {
        // Show progress.
        percentage = (++i / (double)this->dataCount) * 100.0;
        printf("\rOptimizing...%0.4f%%", percentage);

        if (i == 0 || i % dataChunkSize == 0) {
            int nextChunkSize = i + dataChunkSize < dataPointCount ? dataChunkSize : (dataPointCount - i) - 1;

            // Empty the current device vector contents.
            thrust::host_vector<double*>().swap(dataSegment);
            thrust::device_vector<double*>().swap(devDataSegment);
            
            // Copy a chunk (within host memory).
            thrust::copy_n(this->data.begin() + i, nextChunkSize, dataSegment);
            
            // Copy a chunk of data points to the GPU.
            devDataSegment = dataSegment;
        }

        // Backtest all strategies against the current data point.
        optimizer_backtest<<<blockCount, threadsPerBlock>>>(devDataSegment, devStrategies, i % dataChunkSize, configurationCount, investment, profitability);
    }

    // Copy strategies from the GPU back to the host.
    strategies = devStrategies;

    printf("\n");

    // Unload data.
    // TODO
}
